#include "hip/hip_runtime.h"


#include <stdio.h>


__global__ void print_details()
{
    printf("blockIdx.x: %d, blockIdx.y: %d, blockIdx.z: %d, blockDim.x: %d, blockDim.y: %d, blockDim.z: %d, gridDim.x: %d, gridDim.y: %d, gridDim.z: %d \n", 
        blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);

}

int main()
{  
    int nx, ny;
    nx = 16;
    ny = 16;

    dim3 block(8,8);
    dim3 grid(nx/block.x, ny/block.y);

    print_details <<<grid, block>>>();
    hipDeviceSynchronize();

    hipDeviceReset();
    return 0;
}