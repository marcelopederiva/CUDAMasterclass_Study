#include "hip/hip_runtime.h"


#include <stdio.h>


__global__ void hello_cuda()
{
    printf("Hello CUDA world \n");

}

int main()
{   int nx, ny;
    nx = 16;
    ny = 4;
    
    dim3 block(8,2); // Dont need to put (4,1,1) because the other axis will be already initialized with 1
    dim3 grid(nx / block.x , ny / block.y);
    hello_cuda <<<grid,block>>> ();
    hipDeviceSynchronize();

    hipDeviceReset();
    return 0;
}